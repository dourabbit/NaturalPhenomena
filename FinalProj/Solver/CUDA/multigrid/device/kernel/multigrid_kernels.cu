#include "hip/hip_runtime.h"

#ifndef __multigrid_kernels_cu__
#define __multigrid_kernels_cu__

#include"../../common/common.co"

#define __compute_capability__ 10

#if __compute_capability__<=13
#define IM(x,y) __umul24(x,y)
#else
#define IM(x,y) ((x)*(y))
#endif

extern "C"
{

__global__
void kernel_setBC(float* field,
				  const float* BC,
				  const unsigned int xSize,
				  const unsigned int ySize
				  )
{
	const unsigned int tidx=IM(blockDim.x,blockIdx.x)+threadIdx.x;
	unsigned int idx=0;
	if(tidx<xSize){
	    const unsigned int bordT=tidx;
	    const unsigned int bordD=IM(xSize,xSize-1)+tidx;
	    field[bordT]=BC[idx]; idx+=xSize;
	    field[bordD]=BC[idx];
	}
	if(tidx<ySize){
		const unsigned int bordL=IM(xSize,tidx);
		const unsigned int bordR=IM(xSize,tidx)+xSize-1;
		idx+=ySize;
		field[bordL]=BC[idx]; idx+=ySize;
		field[bordR]=BC[idx];
	}
}

__global__
void kernel_residual(float* r,
					 const float* u,
					 const float* d,
					 const unsigned int xSize,
					 const unsigned int ySize,
					 const float qrdx
					 )
{
	__shared__ float cell[DY+2][DX+2];

	const unsigned int globalx=IM(DX,blockIdx.x)+threadIdx.x+1;
	const unsigned int globaly=IM(DY,blockIdx.y)+threadIdx.y+1;
	const unsigned int active=(globalx<xSize-1)&(globaly<ySize-1);
    const unsigned int global=IM(xSize,globaly)+globalx;
	const unsigned int localx=threadIdx.x+1;
	const unsigned int localy=threadIdx.y+1;

	if(active){
		cell[localy][localx]=u[global];
		if(threadIdx.x==0){
			cell[localy][0   ]=u[global-1 ];
			cell[localy][DX+1]=u[global+DX];
		} 
		if(threadIdx.y==0){
			cell[0   ][localx]=u[global-xSize       ];
			cell[DY+1][localx]=u[global+IM(xSize,DY)];
		}
	} __syncthreads();

	if(active){
		r[global]=qrdx*(cell[localy  ][localx-1]+
			            cell[localy  ][localx+1]+
						cell[localy-1][localx  ]+
			            cell[localy+1][localx  ]-
				   4.0f*cell[localy  ][localx  ])-d[global];
		
	}
}
	
__global__
void kernel_relax(float* uDst,
				  const float* uSrc,
				  const float* d,
				  const unsigned int xSize,
				  const unsigned int ySize,
				  const float qdx
				  )
{
    __shared__ float cell[DY+2][DX+2];

	const unsigned int globalx=IM(DX,blockIdx.x)+threadIdx.x+1;
	const unsigned int globaly=IM(DY,blockIdx.y)+threadIdx.y+1;
	const unsigned int active=(globalx<xSize-1)&(globaly<ySize-1);
    const unsigned int global=IM(xSize,globaly)+globalx;
	const unsigned int localx=threadIdx.x+1;
	const unsigned int localy=threadIdx.y+1;

	if(active){
		cell[localy][localx]=uSrc[global];
		if(threadIdx.x==0){
			cell[localy][0   ]=uSrc[global-1 ];
			cell[localy][DX+1]=uSrc[global+DX];
		} 
		if(threadIdx.y==0){
			cell[0   ][localx]=uSrc[global-xSize       ];
			cell[DY+1][localx]=uSrc[global+IM(xSize,DY)];
		}
	} __syncthreads();

	if(active){
		float temp=0.25f*(cell[localy-1][localx  ]+
			              cell[localy+1][localx  ]+
                          cell[localy  ][localx-1]+
			              cell[localy  ][localx+1]-qdx*d[global]);

		uDst[global]=0.66666667f*cell[localy][localx]+0.33333333f*temp;
	}
}

__global__
void kernel_restrict(float* coarse,
					 const float* fine,
					 const unsigned int xCoar,
					 const unsigned int yCoar,
					 const unsigned int xFine,
					 const unsigned int yFine
					 )
{
	__shared__ float cell[DY*2+1][DX*2+2];

	const unsigned int globalx=IM(DX,blockIdx.x)+threadIdx.x+1;
	const unsigned int globaly=IM(DY,blockIdx.y)+threadIdx.y+1;
    const unsigned int active=(globalx<xCoar-1)&(globaly<yCoar-1);

	if(active){
		const unsigned int u=IM(DX*2,blockIdx.x)+threadIdx.x+1;
		const unsigned int v=IM(DY*2,blockIdx.y)+threadIdx.y+1;
		unsigned int idx=IM(xFine,v)+u;
		cell[threadIdx.y   ][threadIdx.x     ]=fine[idx]; idx+=DX;
		cell[threadIdx.y   ][threadIdx.x+DX+1]=fine[idx]; idx+=IM(xFine,DY);
		cell[threadIdx.y+DY][threadIdx.x+DX+1]=fine[idx]; idx-=DX;
		cell[threadIdx.y+DY][threadIdx.x     ]=fine[idx];
	    if(threadIdx.x==0){
			idx=IM(xFine,v)+u+DX*2;
		    cell[threadIdx.y   ][DX*2+1]=fine[idx]; idx+=IM(xFine,DY);
		    cell[threadIdx.y+DY][DX*2+1]=fine[idx];
			cell[threadIdx.y   ][DX    ]=cell[threadIdx.y   ][DX+1];
			cell[threadIdx.y+DY][DX    ]=cell[threadIdx.y+DY][DX+1];
	    }

	    if(threadIdx.y==0){
			idx=IM(xFine,v+DY*2)+u;
		    cell[DY*2][threadIdx.x     ]=fine[idx]; idx+=DX;
		    cell[DY*2][threadIdx.x+DX+1]=fine[idx];
	    }

		if((threadIdx.x==0)&(threadIdx.y==0)){
			idx=IM(xFine,v+DY*2)+u+DX*2;
			cell[DY*2][DX*2+1]=fine[idx];
			cell[DY*2][DX    ]=cell[DY*2][DX+1];
		}

	} __syncthreads();

	if(active){
		const unsigned int localx=(threadIdx.x<<1)+1+(threadIdx.x>=8);
		const unsigned int localy=(threadIdx.y<<1)+1;
        const unsigned int global=IM(xCoar,globaly)+globalx;

		coarse[global]=0.0625f*(cell[localy-1][localx-1] +
			                    cell[localy-1][localx+1] +
							    cell[localy+1][localx-1] +
							    cell[localy+1][localx+1])+
				       0.1250f*(cell[localy-1][localx  ] +
					            cell[localy+1][localx  ] +
						    	cell[localy  ][localx+1] +
							    cell[localy  ][localx+1])+
				       0.2500f*(cell[localy  ][localx  ]);
	}
}

__global__
void kernel_interpc(float* fine,
				    const float* coarse,
				    const unsigned int xFine,
				    const unsigned int yFine,
				    const unsigned int xCoar,
				    const unsigned int yCoar
					)
{
	__shared__ float cell[DY/2+1][DX/2+1];

	if((threadIdx.x<=DY/2)&(threadIdx.y<=DX/2)){
		const unsigned int u=IM(DX/2,blockIdx.x)+threadIdx.x;
		const unsigned int v=IM(DY/2,blockIdx.y)+threadIdx.y;
		const unsigned int idx=IM(xCoar,v)+u;
		cell[threadIdx.y][threadIdx.x]=coarse[idx];
	} __syncthreads();

	const unsigned int globalx=IM(DX,blockIdx.x)+threadIdx.x+1;
	const unsigned int globaly=IM(DY,blockIdx.y)+threadIdx.y+1;
	
	if((globalx<xFine-1)&(globaly<yFine-1)){
		const unsigned int xOdded=(globalx&0x1u)!=0;
		const unsigned int yOdded=(globaly&0x1u)!=0;
		const unsigned int localx=threadIdx.x+1;
		const unsigned int localy=threadIdx.y+1;

		float node;
		if(xOdded){
			if(yOdded){
                node=0.25f*(cell[(localy-1)>>1][(localx-1)>>1]+
                            cell[(localy-1)>>1][(localx+1)>>1]+
                            cell[(localy+1)>>1][(localx-1)>>1]+
                            cell[(localy+1)>>1][(localx+1)>>1]);
			} else {
				node=0.50f*(cell[localy>>1][(localx-1)>>1]+
					        cell[localy>>1][(localx+1)>>1]);
			}
		} else {
			if(yOdded){
				node=0.50f*(cell[(localy-1)>>1][localx>>1]+
					        cell[(localy+1)>>1][localx>>1]);
			} else {
				node=cell[localy>>1][localx>>1];
			}
		}
		
		const unsigned int global=IM(xFine,globaly)+globalx;
		fine[global]-=node;
	}
}

__global__
void kernel_dirichletBC(float* coarse,
						const float* fine,
						const unsigned int xCoar,
						const unsigned int yCoar,
						const unsigned int xFine,
						const unsigned int yFine
						)
{
	const unsigned int tidx=IM(blockDim.x,blockIdx.x)+threadIdx.x;

	if(tidx<xCoar){
	    const unsigned int bordT=tidx;
		const unsigned int bordD=IM(xCoar,xCoar-1)+tidx;
		unsigned int idx=tidx<<1;
		coarse[bordT]=fine[idx]; idx+=IM(xFine,yFine-1);
		coarse[bordD]=fine[idx];
	}
	if(tidx<yCoar){
		const unsigned int bordL=IM(xCoar,tidx);
		const unsigned int bordR=bordL+xCoar-1;
		unsigned int idx=IM(xFine,tidx<<1);
		coarse[bordL]=fine[idx]; idx+=(xFine-1);
		coarse[bordR]=fine[idx];
	}
}

}

#endif